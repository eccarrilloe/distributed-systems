#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <ctime>

#include <opencv2/opencv.hpp>
#include <opencv2/core/core.hpp>
#include <opencv2/videoio.hpp>
#include <opencv2/highgui/highgui.hpp>

using namespace cv;
using namespace cv::cuda;
using namespace std;

VideoCapture openVideo(string filename);
__global__ void processFrame(GpuMat* frames, int* N, double* results);

int main(void)
{
    // Init data and variables
    string filename = "../../samples/sunshine.mp4";
    VideoCapture video = openVideo(filename);

    int frameCount = video.get(CV_CAP_PROP_FRAME_COUNT);
    int samples = 100;
    int interval = frameCount / samples;
    int numBlocks = 4;
    int numThreads = 64;

    printf("[INFO] Setup Data\n");
    int startInitFrames = clock();

    // Get frame samples from VideoCapture
    GpuMat* data = new GpuMat[samples];
    for (int i = 0; i < samples; i++) {
        Mat frame;
        GpuMat gpuFrame;

        video.set(CV_CAP_PROP_POS_FRAMES, i * interval);
        video >> frame;

        gpuFrame.upload(frame);
        data[i] = gpuFrame;
    }

    // Transfer data from Host to Device
    double* results;
    int* N;
    int* d_N;
    N = &samples;

    GpuMat* frames;

    hipMalloc(&d_N, sizeof(int));
    hipMalloc(&frames, samples * sizeof(Mat));
    hipMallocManaged(&results, samples * sizeof(double));

    hipMemcpy((void*) d_N, N, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy((void*) frames, data, samples * sizeof(Mat), hipMemcpyHostToDevice);

    double timeInitFrames = (clock() - startInitFrames) / CLOCKS_PER_SEC;
    printf("[INFO] Time Init: %.2f ms\n", timeInitFrames * 1000);

    printf("[INFO] Running Device Code\n");
    int startProcessingTime = clock();
    processFrame<<<numBlocks, numThreads>>>(frames, d_N, results);

    hipDeviceSynchronize();
    double timeProcessing = (double) (clock() - startProcessingTime) / CLOCKS_PER_SEC;

    for (int i = 0; i < samples; i++) {
        printf("Sample #%i: %.3f\n", i, results[i] / 2.55);
    }

    printf("[INFO] Device Synchronized\n");
    printf("[INFO] Time Processing: %.5f ms\n", timeProcessing * 1000);

    hipFree(data);
    hipFree(results);
    return 0;
}


VideoCapture openVideo(string filename) {
    VideoCapture capture(filename);

    if (!capture.isOpened()) {
        throw "[ERROR] Cannot open filename";
    }

    return capture;
}


__global__ void processFrame(GpuMat* frames, int* N, double* results) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < *N) {
        int rows = frames[index].rows;
        int cols = frames[index].cols;
        int pixels = rows * cols;

        double illumina = 0;
        for (int i = 0; i < rows; i++) {
            for (int j = 0; j < cols; j++) {
                uint8_t pixelR = frames[index].data[i * cols + j * 3 + 0];
                uint8_t pixelG = frames[index].data[i * cols + j * 3 + 1];
                uint8_t pixelB = frames[index].data[i * cols + j * 3 + 2];

                illumina += 0.3 * pixelR + 0.59 * pixelG + 0.11  * pixelB;
            }
        }

        results[index] = illumina / pixels;
    }

}